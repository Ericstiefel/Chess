#include <hip/hip_runtime.h>
#include <cmath>

__global__ void mse_loss_forward_kernel(const float* y_hat, const float* y, float* loss, int N) {
    __shared__ float partial_sums[256];
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int local_idx = threadIdx.x;

    if (global_idx < N) {
        float diff = y_hat[global_idx] - y[global_idx];
        partial_sums[local_idx] = diff * diff;
    } else {
        partial_sums[local_idx] = 0.0f;
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (local_idx < stride) {
            partial_sums[local_idx] += partial_sums[local_idx + stride];
        }
        __syncthreads();
    }

    if (local_idx == 0) {
        loss[blockIdx.x] = partial_sums[0];
    }
}

__global__ void mse_loss_backward_kernel(const float* y_hat, const float* y, float* dL_dy_hat, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        dL_dy_hat[idx] = 2.0f * (y_hat[idx] - y[idx]) / N;
    }
}

extern "C" {
void mse_loss_forward_launcher(const float* y_hat, const float* y, float* loss, int N) {
    dim3 threadsPerBlock(256);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x);
    mse_loss_forward_kernel<<<numBlocks, threadsPerBlock>>>(y_hat, y, loss, N);
}

void mse_loss_backward_launcher(const float* y_hat, const float* y, float* dL_dy_hat, int N) {
    dim3 threadsPerBlock(256);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x);
    mse_loss_backward_kernel<<<numBlocks, threadsPerBlock>>>(y_hat, y, dL_dy_hat, N);
}
}
