#include <hip/hip_runtime.h>
#include <cmath>
#include <limits>

__global__ void softmax_1d_forward_kernel(const float* __restrict__ input, float* __restrict__ output, int size) {
    int tid = threadIdx.x;

    extern __shared__ float sdata[];

    // Step 1: Find the maximum value in the vector (Parallel Reduction)
    float max_val = -std::numeric_limits<float>::infinity();
    for (int j = tid; j < size; j += blockDim.x) {
        max_val = fmaxf(max_val, input[j]);
    }
    sdata[tid] = max_val;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = fmaxf(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }
    max_val = sdata[0];

    // Step 2: Calculate the sum of exponents (Parallel Reduction)
    float sum_val = 0.0f;
    for (int j = tid; j < size; j += blockDim.x) {
        sum_val += expf(input[j] - max_val);
    }
    sdata[tid] = sum_val;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    sum_val = sdata[0];

    // Step 3: Compute the final softmax values
    for (int j = tid; j < size; j += blockDim.x) {
        output[j] = expf(input[j] - max_val) / sum_val;
    }
}

__global__ void softmax_1d_backward_kernel(
    const float* __restrict__ grad_output,
    const float* __restrict__ output,
    float* __restrict__ grad_input,
    int size
) {
    int tid = threadIdx.x;

    extern __shared__ float sdata[];

    // Step 1: Compute dot(grad_output, output)
    float dot_val = 0.0f;
    for (int j = tid; j < size; j += blockDim.x) {
        dot_val += grad_output[j] * output[j];
    }
    sdata[tid] = dot_val;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    dot_val = sdata[0];

    // Step 2: Compute final gradient: grad_input = output * (grad_output - dot_val)
    for (int j = tid; j < size; j += blockDim.x) {
        float s = output[j];
        float dy = grad_output[j];
        grad_input[j] = s * (dy - dot_val);
    }
}

extern "C" {
void softmax_forward_launcher(const float* input, float* output, int size) {
    int threads = 256;
    int blocks = 1; // Always use one block for a 1D vector
    size_t sharedMem = threads * sizeof(float);
    softmax_1d_forward_kernel<<<blocks, threads, sharedMem>>>(input, output, size);
}

void softmax_backward_launcher(const float* grad_output, const float* output, float* grad_input, int size) {
    int threads = 256;
    int blocks = 1; // Always use one block for a 1D vector
    size_t sharedMem = threads * sizeof(float);
    softmax_1d_backward_kernel<<<blocks, threads, sharedMem>>>(grad_output, output, grad_input, size);
}
}